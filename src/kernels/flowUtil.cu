#include "hip/hip_runtime.h"
// System
#include <iostream>
#include <chrono>
#include <string>
#include <stdexcept>

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../common/cuda_helper.h"

// NVIDIA Perf Primitives
#include <nppi.h>
#include <nppi_filtering_functions.h>

#include "../common/timer.h"
#include "../FDF1.0.1/image.h"
#include "flowUtil.h"

using namespace timer;

#define datanorm        0.1f*0.1f      //0.01f // square of the normalization factor
#define epsilon_color  (0.001f*0.001f) //0.000001f
#define epsilon_grad   (0.001f*0.001f) //0.000001f
#define epsilon_desc   (0.001f*0.001f) //0.000001f
#define epsilon_smooth (0.001f*0.001f) //0.000001f

__global__ void kernelDataTerm(
    float *a11c1, float *a12c1, float *a22c1,
    float *b1c1, float *b2c1, 
    float *maskc1, 
    float *wxc1, float *wyc1,
    float *duc1, float *dvc1, 
    float *uuc1, float *vvc1, 
    float *Ixc1,    float *Ixc2,    float *Ixc3,
    float *Iyc1,    float *Iyc2,    float *Iyc3,
    float *Izc1,    float *Izc2,    float *Izc3,
    float *Ixxc1,   float *Ixxc2,   float *Ixxc3,
    float *Ixyc1,   float *Ixyc2,   float *Ixyc3,
    float *Iyyc1,   float *Iyyc2,   float *Iyyc3,
    float *Ixzc1,   float *Ixzc2,   float *Ixzc3,
    float *Iyzc1,   float *Iyzc2,   float *Iyzc3, 
    const float half_delta_over3, const float half_beta, const float half_gamma_over3, int N) {

  int tidx = blockDim.x * blockIdx.x + threadIdx.x;

  if (tidx < N) {

    const float dnorm    = datanorm;
    const float hdover3  = half_delta_over3;
    const float epscolor = epsilon_color;
    const float hgover3  = half_gamma_over3;
    const float epsgrad  = epsilon_grad;

    float *dup  = (float*) duc1 + tidx,
          *dvp = (float*) dvc1 + tidx,
          *maskp = (float*) maskc1 + tidx,
          *a11p  = (float*) a11c1 + tidx,
          *a12p = (float*) a12c1 + tidx,
          *a22p = (float*) a22c1 + tidx, 
          *b1p   = (float*) b1c1 + tidx,
          *b2p = (float*) b2c1 + tidx, 
          *ix1p  = (float*) Ixc1 + tidx,
          *iy1p=(float*)Iyc1 + tidx,
          *iz1p=(float*)Izc1 + tidx,
          *ixx1p=(float*)Ixxc1 + tidx,
          *ixy1p=(float*)Ixyc1 + tidx,
          *iyy1p=(float*)Iyyc1 + tidx,
          *ixz1p=(float*)Ixzc1 + tidx,
          *iyz1p=(float*) Iyzc1 + tidx, 
          *ix2p  = (float*) Ixc2 + tidx,
          *iy2p=(float*)Iyc2 + tidx,
          *iz2p=(float*)Izc2 + tidx,
          *ixx2p=(float*)Ixxc2 + tidx,
          *ixy2p=(float*)Ixyc2 + tidx,
          *iyy2p=(float*)Iyyc2 + tidx,
          *ixz2p=(float*)Ixzc2 + tidx,
          *iyz2p=(float*) Iyzc2 + tidx, 
          *ix3p  = (float*) Ixc3 + tidx,
          *iy3p=(float*)Iyc3 + tidx,
          *iz3p=(float*)Izc3 + tidx,
          *ixx3p=(float*)Ixxc3 + tidx,
          *ixy3p=(float*)Ixyc3 + tidx,
          *iyy3p=(float*)Iyyc3 + tidx,
          *ixz3p=(float*)Ixzc3 + tidx,
          *iyz3p=(float*) Iyzc3 + tidx;


    float tmp, tmp2, n1, n2;
    float tmp3, tmp4, tmp5, tmp6, n3, n4, n5, n6;

    // dpsi color
    if(half_delta_over3){
      tmp  = *iz1p + (*ix1p)*(*dup) + (*iy1p)*(*dvp);
      n1 = (*ix1p) * (*ix1p) + (*iy1p) * (*iy1p) + dnorm;
      tmp2 = *iz2p + (*ix2p)*(*dup) + (*iy2p)*(*dvp);
      n2 = (*ix2p) * (*ix2p) + (*iy2p) * (*iy2p) + dnorm;
      tmp3 = *iz3p + (*ix3p)*(*dup) + (*iy3p)*(*dvp);
      n3 = (*ix3p) * (*ix3p) + (*iy3p) * (*iy3p) + dnorm;
      tmp = (*maskp) * hdover3 / sqrtf(tmp*tmp/n1 + tmp2*tmp2/n2 + tmp3*tmp3/n3 + epscolor);
      tmp3 = tmp/n3; tmp2 = tmp/n2; tmp /= n1;
      *a11p += tmp  * (*ix1p) * (*ix1p);
      *a12p += tmp  * (*ix1p) * (*iy1p);
      *a22p += tmp  * (*iy1p) * (*iy1p);
      *b1p -=  tmp  * (*iz1p) * (*ix1p);
      *b2p -=  tmp  * (*iz1p) * (*iy1p);
      *a11p += tmp2 * (*ix2p) * (*ix2p);
      *a12p += tmp2 * (*ix2p) * (*iy2p);
      *a22p += tmp2 * (*iy2p) * (*iy2p);
      *b1p -=  tmp2 * (*iz2p) * (*ix2p);
      *b2p -=  tmp2 * (*iz2p) * (*iy2p);
      *a11p += tmp3 * (*ix3p) * (*ix3p);
      *a12p += tmp3 * (*ix3p) * (*iy3p);
      *a22p += tmp3 * (*iy3p) * (*iy3p);
      *b1p -=  tmp3 * (*iz3p) * (*ix3p);
      *b2p -=  tmp3 * (*iz3p) * (*iy3p);
    }

    // dpsi gradient
    n1 = (*ixx1p) * (*ixx1p) + (*ixy1p) * (*ixy1p) + dnorm;
    n2 = (*iyy1p) * (*iyy1p) + (*ixy1p) * (*ixy1p) + dnorm;
    tmp  = *ixz1p + (*ixx1p) * (*dup) + (*ixy1p) * (*dvp);
    tmp2 = *iyz1p + (*ixy1p) * (*dup) + (*iyy1p) * (*dvp);
    n3 = (*ixx2p) * (*ixx2p) + (*ixy2p) * (*ixy2p) + dnorm;
    n4 = (*iyy2p) * (*iyy2p) + (*ixy2p) * (*ixy2p) + dnorm;
    tmp3 = *ixz2p + (*ixx2p) * (*dup) + (*ixy2p) * (*dvp);
    tmp4 = *iyz2p + (*ixy2p) * (*dup) + (*iyy2p) * (*dvp);
    n5 = (*ixx3p) * (*ixx3p) + (*ixy3p) * (*ixy3p) + dnorm;
    n6 = (*iyy3p) * (*iyy3p) + (*ixy3p) * (*ixy3p) + dnorm;
    tmp5 = *ixz3p + (*ixx3p) * (*dup) + (*ixy3p) * (*dvp);
    tmp6 = *iyz3p + (*ixy3p) * (*dup) + (*iyy3p) * (*dvp);
    tmp = (*maskp) * hgover3 / sqrtf(
        tmp*tmp/n1 + tmp2*tmp2/n2 + tmp3*tmp3/n3 + tmp4*tmp4/n4 + tmp5*tmp5/n5 + tmp6*tmp6/n6 + epsgrad);
    tmp6 = tmp/n6; tmp5 = tmp/n5; tmp4 = tmp/n4; tmp3 = tmp/n3; tmp2 = tmp/n2; tmp /= n1;      
    *a11p += tmp *(*ixx1p)*(*ixx1p) + tmp2*(*ixy1p)*(*ixy1p);
    *a12p += tmp *(*ixx1p)*(*ixy1p) + tmp2*(*ixy1p)*(*iyy1p);
    *a22p += tmp2*(*iyy1p)*(*iyy1p) + tmp *(*ixy1p)*(*ixy1p);
    *b1p -=  tmp *(*ixx1p)*(*ixz1p) + tmp2*(*ixy1p)*(*iyz1p);
    *b2p -=  tmp2*(*iyy1p)*(*iyz1p) + tmp *(*ixy1p)*(*ixz1p);
    *a11p += tmp3*(*ixx2p)*(*ixx2p) + tmp4*(*ixy2p)*(*ixy2p);
    *a12p += tmp3*(*ixx2p)*(*ixy2p) + tmp4*(*ixy2p)*(*iyy2p);
    *a22p += tmp4*(*iyy2p)*(*iyy2p) + tmp3*(*ixy2p)*(*ixy2p);
    *b1p -=  tmp3*(*ixx2p)*(*ixz2p) + tmp4*(*ixy2p)*(*iyz2p);
    *b2p -=  tmp4*(*iyy2p)*(*iyz2p) + tmp3*(*ixy2p)*(*ixz2p);
    *a11p += tmp5*(*ixx3p)*(*ixx3p) + tmp6*(*ixy3p)*(*ixy3p);
    *a12p += tmp5*(*ixx3p)*(*ixy3p) + tmp6*(*ixy3p)*(*iyy3p);
    *a22p += tmp6*(*iyy3p)*(*iyy3p) + tmp5*(*ixy3p)*(*ixy3p);
    *b1p -=  tmp5*(*ixx3p)*(*ixz3p) + tmp6*(*ixy3p)*(*iyz3p);
    *b2p -=  tmp6*(*iyy3p)*(*iyz3p) + tmp5*(*ixy3p)*(*ixz3p);  
  }

}

__global__ void kernelSubLaplacianVert(
    float *src, float *nextSrc,
    float *dst, float *nextDst,
    float *weights, int height, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tidx < stride) {
    float *wvp    = weights + tidx,
          *srcp   = src + tidx,
          *srcp_s = nextSrc + tidx,
          *dstp   = dst + tidx,
          *dstp_s = nextDst + tidx;

    for (int j = 0; j < height - 1; j++) {
      float tmp = (*wvp) * ((*srcp_s)-(*srcp));
      *dstp += tmp;
      *dstp_s -= tmp;
      wvp += stride; srcp += stride; srcp_s += stride; dstp += stride; dstp_s += stride;
    }
  }

}

__global__ void kernelSubLaplacianHoriz(
    float *src, float *dst, float *weights, float *coeffs, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int col  = tidx % width;

  const int BLOCK_HEIGHT = 1;

  if (tidx < width) {
    float *pSrc         = src + tidx,
          *pDst         = dst + tidx,
          *pWeight      = weights + tidx,
          *pCoeffCalc   = coeffs + tidx,
          *pCoeffUpdate = pCoeffCalc;

    int nBlocks = (height + BLOCK_HEIGHT - 1) / BLOCK_HEIGHT;
    int jCalc = 0;
    int jUpdate = 0;

    // Block calculation and update so coeffs fit in cache

    for (int iBlock = 0; iBlock < nBlocks; iBlock++) {

      // Calc coeffs
      for (int j = 0; j < BLOCK_HEIGHT && jCalc < height; j++, jCalc++) {
        // Do not calculate the last column
        if (col != width - 1)
          *pCoeffCalc = (*pWeight) * ( *(pSrc + 1) - *pSrc );

        pSrc += stride; pWeight += stride; pCoeffCalc += stride;
      }

      // Update dst
      for (int j = 0; j < BLOCK_HEIGHT && jUpdate < height; j++, jUpdate++) {
        float update = 0.0;

        if (col != 0)
          update -= *(pCoeffUpdate - 1);
        if (col != width - 1)
          update += *pCoeffUpdate;

        *pDst += update;

        pDst += stride; pCoeffUpdate += stride;
      }
    }
  }
}

__global__ void kernelSubLaplacianHorizFillCoeffs(
    float *src, float *weights, float *coeffs, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int col  = tidx % stride;

  // Do not calculate the last column
  if (tidx < width && col != width - 1) {
    float *pSrc    = src + tidx,
          *pWeight = weights + tidx,
          *pCoeff  = coeffs + tidx;

    for (int j = 0; j < height; j++) {
      *pCoeff = (*pWeight) * ( *(pSrc + 1) - *pSrc );

      pSrc += stride; pWeight += stride; pCoeff += stride;
    }
  }
}

__global__ void kernelSubLaplacianHorizApplyCoeffs(
    float *dst, float *coeffs, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int col  = tidx % stride;

  if (tidx < width) {

    float *pDst   = dst + tidx,
          *pCoeff = coeffs + tidx;

    for (int j = 0; j < height; j++) {
      float update = 0.0;

      if (col != 0)
        update -= *(pCoeff - 1);
      if (col != width - 1)
        update += *pCoeff;

      *pDst += update;

      pDst += stride; pCoeff += stride;
    }
  }

  // if (col < width) {

  //   float *pDst   = dst + tidx,
  //         *pCoeff = coeffs + tidx;

  //   float update = 0.0;

  //   if (col != 0)
  //     update -= *(pCoeff - 1);
  //   if (col != width - 1)
  //     update += *pCoeff;

  //   *pDst += update;
  // }
}

__global__ void kernelSorStep(
    float *du, float *dv,
    float *a11, float *a12, float *a22,
    const float *b1, const float *b2,
    const float *horiz, const float *vert,
    const int iterations, const float omega,
    int height, int width, int stride, bool odd) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int j  = tidx / width;
  int i  = tidx % width;

  bool shouldRun = (odd)
    ? ((i + j) % 2 == 1)
    : ((i + j) % 2 == 0);

  if (tidx < width * height && shouldRun) {

    float sigma_u,sigma_v,sum_dpsis,A11,A22,A12,B1,B2;
    sigma_u = 0.0f;
    sigma_v = 0.0f;
    sum_dpsis = 0.0f;

    int here  = j * stride + i;
    int left  = j * stride + i - 1;
    int right = j * stride + i + 1;
    int up    = (j-1) * stride + i;
    int down  = (j+1) * stride + i;

    if(j>0)
    {
      sigma_u   -= vert[up] * du[up];
      sigma_v   -= vert[up] * dv[up];
      sum_dpsis += vert[up];
    }
    if(i>0)
    {
      sigma_u   -= horiz[left] * du[left];
      sigma_v   -= horiz[left] * dv[left];
      sum_dpsis += horiz[left];
    }
    if(j<height-1)
    {
      sigma_u   -= vert[here] * du[down];
      sigma_v   -= vert[here] * dv[down];
      sum_dpsis += vert[here];
    }
    if(i<width-1)
    {
      sigma_u   -= horiz[here] * du[right];
      sigma_v   -= horiz[here] * dv[right];
      sum_dpsis += horiz[here];
    }

    A11 = a11[here] + sum_dpsis;
    A12 = a12[here];
    A22 = a22[here] + sum_dpsis;

    B1 = b1[here] - sigma_u;
    B2 = b2[here] - sigma_v;

    du[here] = (1.0f-omega) * du[here] + omega/A11 * (B1 - A12 * dv[here]);
    dv[here] = (1.0f-omega) * dv[here] + omega/A22 * (B2 - A12 * du[here]);

  }
}

__global__ void kernelGetMeanImageAndDiff(
    float *img1, float *img2, float *avgImg, float *diff,
    int height, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  // For 3 channels images
  if (tidx < 3 * stride) {
    float
      *pImg1   = img1 + tidx,
      *pImg2   = img2 + tidx,
      *pAvgImg = avgImg + tidx,
      *pDiff   = diff + tidx;

    for (int j = 0; j < height; j++) {
      *pAvgImg = 0.5 * ((*pImg1) + (*pImg2));
      *pDiff   = (*pImg2) - (*pImg1);

      pImg1 += stride; pImg2 += stride; pAvgImg += stride; pDiff += stride;
    }

  }

}


__global__ void kernelFlowMag(
    float *dst,  float *ux,  float *uy,  float *vx,  float *vy,
    float qa, float epsmooth, int height, int width, int stride, int N) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  for (; i < N; i+= blockDim.x * gridDim.x) {
    dst[i] = qa / sqrt(
        (ux[i])*(ux[i]) + (uy[i])*(uy[i]) + (vx[i])*(vx[i]) + (vy[i])*(vy[i]) + epsmooth );
  }

  // if (tidx < stride) {
  //   float *uxp = ux + tidx,
  //         *uyp = uy + tidx,
  //         *vxp = vx + tidx,
  //         *vyp = vy + tidx,
  //         *sp  = dst + tidx;

  //   for (int j = 0; j < height; j++) {
  //     *sp = qa / sqrtf(
  //         (*uxp)*(*uxp) + (*uyp)*(*uyp) + (*vxp)*(*vxp) + (*vyp)*(*vyp) + epsmooth );

  //     uxp += stride; uyp += stride; vxp += stride; vyp += stride; sp += stride; 
  //   }
  // }
}

__global__ void kernelSmoothnessHorizVert(
    float *dst_horiz, float *dst_vert, float *smoothness, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tidx < height * stride) {

    float *dst_horiz_p = dst_horiz + tidx,
          *dst_vert_p  = dst_vert  + tidx,
          *sp          = smoothness + tidx;

    *dst_horiz_p = *sp + *(sp + 1);
    *dst_vert_p  = *sp + *(sp + stride);
  }
}


__global__ void kernelFlowUpdate(
    float *uu, float *vv, float *wx, float *wy, float *du, float *dv,
    int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tidx < height * stride) {

    float *uup = uu + tidx,
          *vvp = vv + tidx,
          *wxp = wx + tidx,
          *wyp = wy + tidx,
          *dup = du + tidx,
          *dvp = dv + tidx;

    (*uup) = (*wxp) + (*dup);
    (*vvp) = (*wyp) + (*dvp);
  }

}


__global__ void kernelWarpImage(
    float *dst1, float *dst2, float *dst3, float *mask,
    float *src1, float *src2, float *src3,
    float *wx, float *wy,
    int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  int i = tidx % stride;
  int j = tidx / stride;
  int offset = j * stride + i;

  if (i < width && j < height) {
    float xx = i + wx[offset];
    float yy = j + wy[offset];
    int x = floor(xx);
    int y = floor(yy);
    float dx = xx - x;
    float dy = yy - y;

    // Set mask according to bounds
    mask[offset] = (xx >= 0 && xx < width && yy >= 0 && yy < height);

    int x1 = MINMAX_TA(x, width);
    int x2 = MINMAX_TA(x + 1, width);
    int y1 = MINMAX_TA(y, height);
    int y2 = MINMAX_TA(y + 1, height);

    dst1[offset] = 
      src1[y1 * stride + x1] * (1.0f-dx) * (1.0f-dy) +
      src1[y1 * stride + x2] * dx * (1.0f-dy) +
      src1[y2 * stride + x1] * (1.0f-dx) * dy +
      src1[y2 * stride + x2] * dx * dy;
    dst2[offset] = 
      src2[y1 * stride + x1] * (1.0f-dx) * (1.0f-dy) +
      src2[y1 * stride + x2] * dx * (1.0f-dy) +
      src2[y2 * stride + x1] * (1.0f-dx) * dy +
      src2[y2 * stride + x2] * dx * dy;
    dst3[offset] = 
      src3[y1 * stride + x1] * (1.0f-dx) * (1.0f-dy) +
      src3[y1 * stride + x2] * dx * (1.0f-dy) +
      src3[y2 * stride + x1] * (1.0f-dx) * dy +
      src3[y2 * stride + x2] * dx * dy;

  }
}


__global__ void kernelSepFlow(
    float *flowx, float *flowy, float *flowout, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  int ix = tidx % width;
  int iy = tidx / width;

  if (ix < width && iy < height) {
    
    int i = iy * width + ix;
    int is = iy * stride + ix;

    flowx[is] = flowout[2 * i];
    flowy[is] = flowout[2 * i + 1];
  }
}


__global__ void kernelMergeFlow(
    float *flowx, float *flowy, float *flowout, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  int ix = tidx % width;
  int iy = tidx / width;

  if (ix < width && iy < height) {
    
    int i = iy * width + ix;
    int is = iy * stride + ix;

    flowout[2 * i] = flowx[is];
    flowout[2 * i + 1] = flowy[is];
  }
}

__global__ void kernelCopyImage(
    float *dst1, float *dst2, float *dst3, const float *src,
    int width_pad, int padding, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  const float *pSrcStart = src + 3 * (width_pad + 1) * padding;

  int ix = tidx % width;
  int iy = tidx / width;

  if (ix < width && iy < height) {

    const float *pSrc = pSrcStart + (iy * 3 * width_pad) + 3 * ix;
    int i = iy * stride + ix;

    dst1[i] = *pSrc; pSrc++;
    dst2[i] = *pSrc; pSrc++;
    dst3[i] = *pSrc; pSrc++;
  }
}


namespace cu {

  void dataTerm(
      image_t *a11, image_t *a12, image_t *a22,
      image_t *b1, image_t *b2, 
      image_t *mask, 
      image_t *wx, image_t *wy,
      image_t *du, image_t *dv, 
      image_t *uu, image_t *vv, 
      color_image_t *Ix,  color_image_t *Iy,  color_image_t *Iz,
      color_image_t *Ixx, color_image_t *Ixy, color_image_t *Iyy,
      color_image_t *Ixz, color_image_t *Iyz, 
      const float half_delta_over3, const float half_beta, const float half_gamma_over3) {

    checkCudaErrors( hipMemset(a11->c1, 0, sizeof(float)*uu->height*uu->stride) );
    checkCudaErrors( hipMemset(a12->c1, 0, sizeof(float)*uu->height*uu->stride) );
    checkCudaErrors( hipMemset(a22->c1, 0, sizeof(float)*uu->height*uu->stride) );
    checkCudaErrors( hipMemset(b1->c1 , 0, sizeof(float)*uu->height*uu->stride) );
    checkCudaErrors( hipMemset(b2->c1 , 0, sizeof(float)*uu->height*uu->stride) );

    // Set up device pointers
    float *a11c1,
          *a12c1,    *a22c1,
          *b1c1,     *b2c1, 
          *maskc1, 
          *wxc1,     *wyc1,
          *duc1,     *dvc1, 
          *uuc1,     *vvc1, 
          *Ixc1,     *Ixc2,     *Ixc3,
          *Iyc1,     *Iyc2,     *Iyc3,
          *Izc1,     *Izc2,     *Izc3,
          *Ixxc1,    *Ixxc2,    *Ixxc3,
          *Ixyc1,    *Ixyc2,    *Ixyc3,
          *Iyyc1,    *Iyyc2,    *Iyyc3,
          *Ixzc1,    *Ixzc2,    *Ixzc3,
          *Iyzc1,    *Iyzc2,    *Iyzc3;

    a11c1  =  a11->c1;
    a12c1  =  a12->c1;
    a22c1  =  a22->c1;
    b1c1   =  b1->c1;
    b2c1   =  b2->c1;
    maskc1 =  mask->c1;
    wxc1   =  wx->c1;
    wyc1   =  wy->c1;
    duc1   =  du->c1;
    dvc1   =  dv->c1;
    uuc1   =  uu->c1;
    vvc1   =  vv->c1;
    Ixc1   =  Ix->c1;
    Ixc2   =  Ix->c2;
    Ixc3   =  Ix->c3;
    Iyc1   =  Iy->c1;
    Iyc2   =  Iy->c2;
    Iyc3   =  Iy->c3;
    Izc1   =  Iz->c1;
    Izc2   =  Iz->c2;
    Izc3   =  Iz->c3;
    Ixxc1  =  Ixx->c1;
    Ixxc2  =  Ixx->c2;
    Ixxc3  =  Ixx->c3;
    Ixyc1  =  Ixy->c1;
    Ixyc2  =  Ixy->c2;
    Ixyc3  =  Ixy->c3;
    Iyyc1  =  Iyy->c1;
    Iyyc2  =  Iyy->c2;
    Iyyc3  =  Iyy->c3;
    Ixzc1  =  Ixz->c1;
    Ixzc2  =  Ixz->c2;
    Ixzc3  =  Ixz->c3;
    Iyzc1  =  Iyz->c1;
    Iyzc2  =  Iyz->c2;
    Iyzc3  =  Iyz->c3;

    int N = uu->height*uu->stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelDataTerm<<<nBlocks, nThreadsPerBlock>>>(
        a11c1, a12c1, a22c1,
        b1c1, b2c1, 
        maskc1, 
        wxc1, wyc1,
        duc1, dvc1, 
        uuc1, vvc1, 
        Ixc1,    Ixc2,    Ixc3,
        Iyc1,    Iyc2,    Iyc3,
        Izc1,    Izc2,    Izc3,
        Ixxc1,   Ixxc2,   Ixxc3,
        Ixyc1,   Ixyc2,   Ixyc3,
        Iyyc1,   Iyyc2,   Iyyc3,
        Ixzc1,   Ixzc2,   Ixzc3,
        Iyzc1,   Iyzc2,   Iyzc3, 
        half_delta_over3, half_beta, half_gamma_over3, N);

  };

  void subLaplacian(
      image_t *dst, const image_t *src, const image_t *weight_horiz, const image_t *weight_vert, float *coeffs) {

    cu::subLaplacianHoriz(src->c1, dst->c1, weight_horiz->c1, coeffs, src->height, src->width, src->stride);
    cu::subLaplacianVert(src->c1, dst->c1, weight_vert->c1, src->height, src->stride);

  }

  void subLaplacianHoriz(
      float *src, float *dst, float *weights, float *coeffs, int height, int width, int stride) {

    float *pDeviceCoeffs = coeffs;

    float *pDeviceSrc = src,
          *pDeviceDst = dst,
          *pDeviceWeights = weights;

    int N = width;
    // int N = height * stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    auto start_horiz = now();

    kernelSubLaplacianHorizFillCoeffs<<<nBlocks, nThreadsPerBlock>>>(
        pDeviceSrc, pDeviceWeights, pDeviceCoeffs, height, width, stride);

    kernelSubLaplacianHorizApplyCoeffs<<<nBlocks, nThreadsPerBlock>>>(
        pDeviceDst, pDeviceCoeffs, height, width, stride);

    // kernelSubLaplacianHoriz<<<nBlocks, nThreadsPerBlock>>>(
    //     pDeviceSrc, pDeviceDst, pDeviceWeights, pDeviceCoeffs, height, width, stride);
    calc_print_elapsed("laplacian horiz", start_horiz);
  }

  void subLaplacianVert(
      float *src, float *dst, float *weights, int height, int stride) {

    int N = stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    auto start_vert = now();
    kernelSubLaplacianVert<<<nBlocks, nThreadsPerBlock>>>(
        src, src + stride, dst, dst + stride, weights, height, stride);
    calc_print_elapsed("laplacian vert", start_vert);
  }

  void sor(
      float *du, float *dv,
      float *a11, float *a12, float *a22,
      float *b1, float *b2,
      float *horiz, float *vert,
      int iterations, float omega,
      int height, int width, int stride) {

    // Device setup
    float 
      *d_du,
    *d_dv,
    *d_a11,
    *d_a12,
    *d_a22,
    *d_b1,
    *d_b2,
    *d_horiz,
    *d_vert;

    d_du    = du;
    d_dv    = dv;
    d_a11   = a11;
    d_a12   = a12;
    d_a22   = a22;
    d_b1    = b1;
    d_b2    = b2;
    d_horiz = horiz;
    d_vert  = vert;

    int N = width * height;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    for(int iter = 0 ; iter<iterations ; iter++)
    {

      auto start_sor_odd = now();
      kernelSorStep<<<nBlocks, nThreadsPerBlock>>>(
          d_du, d_dv,
          d_a11, d_a12, d_a22,
          d_b1, d_b2,
          d_horiz, d_vert,
          iterations, omega,
          height, width, stride, true);

      hipDeviceSynchronize();
      calc_print_elapsed("sor step odd", start_sor_odd);

      auto start_sor_even = now();
      kernelSorStep<<<nBlocks, nThreadsPerBlock>>>(
          d_du, d_dv,
          d_a11, d_a12, d_a22,
          d_b1, d_b2,
          d_horiz, d_vert,
          iterations, omega,
          height, width, stride, false);
      hipDeviceSynchronize();
      calc_print_elapsed("sor step even", start_sor_even);
    }  
  }

  void getMeanImageAndDiff(
      float *img1, float *img2, float *avgImg, float *diff,
      int height, int stride) {

    int N = 3 * stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelGetMeanImageAndDiff<<<nBlocks, nThreadsPerBlock>>>(
        img1, img2, avgImg, diff,
        height, stride);

  }

  void colorImageDerivative(
      float *dst, float *src, float *pDeviceColorDerivativeKernel, 
      int height, int width, int stride, bool horiz) {

    Npp32f *pDeviceSrc = src;
    Npp32f *pDeviceDst = dst;

    size_t elemSize = sizeof(float);
    unsigned int nSrcStep = stride * elemSize;
    unsigned int nDstStep = nSrcStep;

    NppiSize oSrcSize = { width, height };
    NppiPoint oSrcOffset = { 0, 0 };
    NppiSize oSizeROI = { width, height };
    NppiBorderType eBorderType = NPP_BORDER_REPLICATE;

    NPP_CHECK_NPP(
        (horiz)
        ? nppiFilterRowBorder_32f_C1R (
          pDeviceSrc, nSrcStep, oSrcSize, oSrcOffset,
          pDeviceDst, nDstStep, oSizeROI,
          pDeviceColorDerivativeKernel, 5, 2, eBorderType)
        : nppiFilterColumnBorder_32f_C1R (
          pDeviceSrc, nSrcStep, oSrcSize, oSrcOffset,
          pDeviceDst, nDstStep, oSizeROI,
          pDeviceColorDerivativeKernel, 5, 2, eBorderType)
        );
  }

  // Expects filter kernel of the form
  //   { -0.5, 0.0, 0.5 }
  void imageDerivative(
      float *dst, float *src, float *pDeviceDerivativeKernel, 
      int height, int width, int stride, bool horiz) {

    Npp32f *pDeviceSrc = src;
    Npp32f *pDeviceDst = dst;

    size_t elemSize = sizeof(float);
    unsigned int nSrcStep = stride * elemSize;
    unsigned int nDstStep = nSrcStep;

    NppiSize oSrcSize = { width, height };
    NppiPoint oSrcOffset = { 0, 0 };
    NppiSize oSizeROI = { width, height };
    NppiBorderType eBorderType = NPP_BORDER_REPLICATE;

    NPP_CHECK_NPP(
        (horiz)
        ? nppiFilterRowBorder_32f_C1R (
          pDeviceSrc, nSrcStep, oSrcSize, oSrcOffset,
          pDeviceDst, nDstStep, oSizeROI,
          pDeviceDerivativeKernel, 3, 1, eBorderType)
        : nppiFilterColumnBorder_32f_C1R (
          pDeviceSrc, nSrcStep, oSrcSize, oSrcOffset,
          pDeviceDst, nDstStep, oSizeROI,
          pDeviceDerivativeKernel, 3, 1, eBorderType)
        );
  }

  void smoothnessTerm(
      float *dst_horiz, float *dst_vert, float *smoothness,
      float *ux,  float *uy,  float *vx,  float *vy,
      float qa, float epsmooth,
      int height, int width, int stride) {

    int N = height * width;
    int nThreadsPerBlock = 64;
    int nBlocks = 56;

    auto start_mag = now();
    kernelFlowMag<<<nBlocks, nThreadsPerBlock>>> (
        smoothness, ux, uy, vx, vy,
        qa, epsmooth, height, width, stride, N);
    hipDeviceSynchronize();
    calc_print_elapsed("smoothnessTerm magnitude", start_mag);

    N = height * stride;
    nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    auto start_horizvert = now();
    kernelSmoothnessHorizVert<<< nBlocks, nThreadsPerBlock >>> (
        dst_horiz, dst_vert, smoothness, height, width, stride);
    hipDeviceSynchronize();
    calc_print_elapsed("smoothnessTerm horiz vert", start_horizvert);

  }


  void flowUpdate(
      float *uu, float *vv, float *wx, float *wy, float *du, float *dv,
      int height, int width, int stride) {

    int N = height * stride;
    int nThreadsPerBlock = 128;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelFlowUpdate<<< nBlocks, nThreadsPerBlock >>> (
        uu, vv, wx, wy, du, dv,
        height, width, stride);

  }

  /*
     Warp an image `src` into `dst` using warp vectors `wx`, `wy`.
     Store `mask[i]` = 0 or 1 if pixel i goes outisde or inside image bounds.
   */
  void warpImage(
      color_image_t *dst, image_t *mask, const color_image_t *src, const image_t *wx, const image_t *wy) {

    int N = src->height * src->stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelWarpImage<<< nBlocks, nThreadsPerBlock >>> (
        dst->c1, dst->c2, dst->c3, mask->c1,
        src->c1, src->c2, src->c3,
        wx->c1,  wy->c1,  src->height, src->width, src->stride);
  }

  
  void computeSmoothness(
      image_t *dst_horiz, image_t *dst_vert, const image_t *uu, const image_t *vv, float *deriv_flow,
      image_t *ux, image_t *uy, image_t *vx, image_t *vy, image_t *smoothness, 
      const float quarter_alpha) {

    auto start_setup = now();
    const int width = uu->width, height = vv->height, stride = uu->stride;
    calc_print_elapsed("smoothness setup", start_setup);

    // compute derivatives [-0.5 0 0.5]
    auto start_derivs = now();
    cu::imageDerivative(ux->c1, uu->c1, deriv_flow, height, width, stride, true);
    cu::imageDerivative(vx->c1, vv->c1, deriv_flow, height, width, stride, true);
    cu::imageDerivative(uy->c1, uu->c1, deriv_flow, height, width, stride, false);
    cu::imageDerivative(vy->c1, vv->c1, deriv_flow, height, width, stride, false);
    calc_print_elapsed("smoothness derivatives", start_derivs);

    auto start_calc = now();
    cu::smoothnessTerm(
        dst_horiz->c1, dst_vert->c1, smoothness->c1,
        ux->c1, uy->c1, vx->c1, vy->c1,
        quarter_alpha, epsilon_smooth,
        height, width, stride);
    calc_print_elapsed("smoothness term", start_calc);

    // Cleanup extra columns
    auto start_cleanup = now();
    // // Doesn't really affect the output
    // for(int j = 0; j < height; j++){
    //   // memset(&dst_horiz->c1[j*stride+width-1], 0, sizeof(float)*(stride-width+1));
    //   checkCudaErrors( hipMemset(&dst_horiz->c1[j*stride+width-1], 0, sizeof(float)*(stride-width+1)) );
    // }
    checkCudaErrors( hipMemset( &dst_vert->c1[(height-1)*stride], 0, sizeof(float)*stride) );
    calc_print_elapsed("smoothness cleanup", start_cleanup);
  }

  void getDerivatives(
      const color_image_t *im1, const color_image_t *im2, float *pDeviceKernel,
      color_image_t *dx, color_image_t *dy, color_image_t *dt, 
      color_image_t *dxx, color_image_t *dxy, color_image_t *dyy, color_image_t *dxt, color_image_t *dyt)
  {
    // derivatives are computed on the mean of the first image and the warped second image
    color_image_t *tmp_im2 = color_image_new(im2->width,im2->height);    

    int height = im2->height;
    int width = im2->width;
    int stride = im2->stride;

    cu::getMeanImageAndDiff(im1->c1, im2->c1, tmp_im2->c1, dt->c1, im1->height, im1->stride);

    // compute all other derivatives
    cu::colorImageDerivative(dx->c1,  tmp_im2->c1, pDeviceKernel, height, width, stride, true); // horizontal
    cu::colorImageDerivative(dy->c1,  tmp_im2->c1, pDeviceKernel, height, width, stride, false);
    cu::colorImageDerivative(dxx->c1, dx->c1,      pDeviceKernel, height, width, stride, true);
    cu::colorImageDerivative(dxy->c1, dx->c1,      pDeviceKernel, height, width, stride, false);
    cu::colorImageDerivative(dyy->c1, dy->c1,      pDeviceKernel, height, width, stride, false);
    cu::colorImageDerivative(dxt->c1, dt->c1,      pDeviceKernel, height, width, stride, true);
    cu::colorImageDerivative(dyt->c1, dt->c1,      pDeviceKernel, height, width, stride, false);

    // free memory
    color_image_delete(tmp_im2);
  }

  void sepFlow(std::vector<image_t*> flow_sep, float *flowout, int height, int width) {

    int N = width * height;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelSepFlow<<< nBlocks, nThreadsPerBlock >>>(
        flow_sep[0]->c1, flow_sep[1]->c1, flowout, height, width, flow_sep[0]->stride);

  }

  void mergeFlow(std::vector<image_t*> flow_sep, float *flowout, int height, int width) {

    int N = width * height;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelMergeFlow<<< nBlocks, nThreadsPerBlock >>>(
        flow_sep[0]->c1, flow_sep[1]->c1, flowout, height, width, flow_sep[0]->stride);
  }

  void copyImage(color_image_t *dst, const float *src, int width_pad, int padding, int height, int width) {

    int N = width * height;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelCopyImage<<< nBlocks, nThreadsPerBlock >>> (
        dst->c1, dst->c2, dst->c3, src,
        width_pad, padding, height, width, dst->stride);
  }

}
