#include "hip/hip_runtime.h"
// System
#include <iostream>
#include <chrono>
#include <string>
#include <stdexcept>

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// NVIDIA Perf Primitives
#include <nppi.h>
#include <nppi_filtering_functions.h>

#include "../common/timer.h"
#include "../FDF1.0.1/image.h"
#include "flowUtil.h"

using namespace timer;

#define datanorm        0.1f*0.1f      //0.01f // square of the normalization factor
#define epsilon_color  (0.001f*0.001f) //0.000001f
#define epsilon_grad   (0.001f*0.001f) //0.000001f
#define epsilon_desc   (0.001f*0.001f) //0.000001f
#define epsilon_smooth (0.001f*0.001f) //0.000001f

__global__ void kernelDataTerm(
    float *a11c1, float *a12c1, float *a22c1,
    float *b1c1, float *b2c1, 
    float *maskc1, 
    float *wxc1, float *wyc1,
    float *duc1, float *dvc1, 
    float *uuc1, float *vvc1, 
    float *Ixc1,    float *Ixc2,    float *Ixc3,
    float *Iyc1,    float *Iyc2,    float *Iyc3,
    float *Izc1,    float *Izc2,    float *Izc3,
    float *Ixxc1,   float *Ixxc2,   float *Ixxc3,
    float *Ixyc1,   float *Ixyc2,   float *Ixyc3,
    float *Iyyc1,   float *Iyyc2,   float *Iyyc3,
    float *Ixzc1,   float *Ixzc2,   float *Ixzc3,
    float *Iyzc1,   float *Iyzc2,   float *Iyzc3, 
    const float half_delta_over3, const float half_beta, const float half_gamma_over3, int N) {

  int tidx = blockDim.x * blockIdx.x + threadIdx.x;

  if (tidx < N) {

    const float dnorm    = datanorm;
    const float hdover3  = half_delta_over3;
    const float epscolor = epsilon_color;
    const float hgover3  = half_gamma_over3;
    const float epsgrad  = epsilon_grad;

    float *dup  = (float*) duc1 + tidx,
          *dvp = (float*) dvc1 + tidx,
          *maskp = (float*) maskc1 + tidx,
          *a11p  = (float*) a11c1 + tidx,
          *a12p = (float*) a12c1 + tidx,
          *a22p = (float*) a22c1 + tidx, 
          *b1p   = (float*) b1c1 + tidx,
          *b2p = (float*) b2c1 + tidx, 
          *ix1p  = (float*) Ixc1 + tidx,
          *iy1p=(float*)Iyc1 + tidx,
          *iz1p=(float*)Izc1 + tidx,
          *ixx1p=(float*)Ixxc1 + tidx,
          *ixy1p=(float*)Ixyc1 + tidx,
          *iyy1p=(float*)Iyyc1 + tidx,
          *ixz1p=(float*)Ixzc1 + tidx,
          *iyz1p=(float*) Iyzc1 + tidx, 
          *ix2p  = (float*) Ixc2 + tidx,
          *iy2p=(float*)Iyc2 + tidx,
          *iz2p=(float*)Izc2 + tidx,
          *ixx2p=(float*)Ixxc2 + tidx,
          *ixy2p=(float*)Ixyc2 + tidx,
          *iyy2p=(float*)Iyyc2 + tidx,
          *ixz2p=(float*)Ixzc2 + tidx,
          *iyz2p=(float*) Iyzc2 + tidx, 
          *ix3p  = (float*) Ixc3 + tidx,
          *iy3p=(float*)Iyc3 + tidx,
          *iz3p=(float*)Izc3 + tidx,
          *ixx3p=(float*)Ixxc3 + tidx,
          *ixy3p=(float*)Ixyc3 + tidx,
          *iyy3p=(float*)Iyyc3 + tidx,
          *ixz3p=(float*)Ixzc3 + tidx,
          *iyz3p=(float*) Iyzc3 + tidx;


    float tmp, tmp2, n1, n2;
    float tmp3, tmp4, tmp5, tmp6, n3, n4, n5, n6;

    // dpsi color
    if(half_delta_over3){
      tmp  = *iz1p + (*ix1p)*(*dup) + (*iy1p)*(*dvp);
      n1 = (*ix1p) * (*ix1p) + (*iy1p) * (*iy1p) + dnorm;
      tmp2 = *iz2p + (*ix2p)*(*dup) + (*iy2p)*(*dvp);
      n2 = (*ix2p) * (*ix2p) + (*iy2p) * (*iy2p) + dnorm;
      tmp3 = *iz3p + (*ix3p)*(*dup) + (*iy3p)*(*dvp);
      n3 = (*ix3p) * (*ix3p) + (*iy3p) * (*iy3p) + dnorm;
      tmp = (*maskp) * hdover3 / sqrtf(tmp*tmp/n1 + tmp2*tmp2/n2 + tmp3*tmp3/n3 + epscolor);
      tmp3 = tmp/n3; tmp2 = tmp/n2; tmp /= n1;
      *a11p += tmp  * (*ix1p) * (*ix1p);
      *a12p += tmp  * (*ix1p) * (*iy1p);
      *a22p += tmp  * (*iy1p) * (*iy1p);
      *b1p -=  tmp  * (*iz1p) * (*ix1p);
      *b2p -=  tmp  * (*iz1p) * (*iy1p);
      *a11p += tmp2 * (*ix2p) * (*ix2p);
      *a12p += tmp2 * (*ix2p) * (*iy2p);
      *a22p += tmp2 * (*iy2p) * (*iy2p);
      *b1p -=  tmp2 * (*iz2p) * (*ix2p);
      *b2p -=  tmp2 * (*iz2p) * (*iy2p);
      *a11p += tmp3 * (*ix3p) * (*ix3p);
      *a12p += tmp3 * (*ix3p) * (*iy3p);
      *a22p += tmp3 * (*iy3p) * (*iy3p);
      *b1p -=  tmp3 * (*iz3p) * (*ix3p);
      *b2p -=  tmp3 * (*iz3p) * (*iy3p);
    }

    // dpsi gradient
    n1 = (*ixx1p) * (*ixx1p) + (*ixy1p) * (*ixy1p) + dnorm;
    n2 = (*iyy1p) * (*iyy1p) + (*ixy1p) * (*ixy1p) + dnorm;
    tmp  = *ixz1p + (*ixx1p) * (*dup) + (*ixy1p) * (*dvp);
    tmp2 = *iyz1p + (*ixy1p) * (*dup) + (*iyy1p) * (*dvp);
    n3 = (*ixx2p) * (*ixx2p) + (*ixy2p) * (*ixy2p) + dnorm;
    n4 = (*iyy2p) * (*iyy2p) + (*ixy2p) * (*ixy2p) + dnorm;
    tmp3 = *ixz2p + (*ixx2p) * (*dup) + (*ixy2p) * (*dvp);
    tmp4 = *iyz2p + (*ixy2p) * (*dup) + (*iyy2p) * (*dvp);
    n5 = (*ixx3p) * (*ixx3p) + (*ixy3p) * (*ixy3p) + dnorm;
    n6 = (*iyy3p) * (*iyy3p) + (*ixy3p) * (*ixy3p) + dnorm;
    tmp5 = *ixz3p + (*ixx3p) * (*dup) + (*ixy3p) * (*dvp);
    tmp6 = *iyz3p + (*ixy3p) * (*dup) + (*iyy3p) * (*dvp);
    tmp = (*maskp) * hgover3 / sqrtf(
        tmp*tmp/n1 + tmp2*tmp2/n2 + tmp3*tmp3/n3 + tmp4*tmp4/n4 + tmp5*tmp5/n5 + tmp6*tmp6/n6 + epsgrad);
    tmp6 = tmp/n6; tmp5 = tmp/n5; tmp4 = tmp/n4; tmp3 = tmp/n3; tmp2 = tmp/n2; tmp /= n1;      
    *a11p += tmp *(*ixx1p)*(*ixx1p) + tmp2*(*ixy1p)*(*ixy1p);
    *a12p += tmp *(*ixx1p)*(*ixy1p) + tmp2*(*ixy1p)*(*iyy1p);
    *a22p += tmp2*(*iyy1p)*(*iyy1p) + tmp *(*ixy1p)*(*ixy1p);
    *b1p -=  tmp *(*ixx1p)*(*ixz1p) + tmp2*(*ixy1p)*(*iyz1p);
    *b2p -=  tmp2*(*iyy1p)*(*iyz1p) + tmp *(*ixy1p)*(*ixz1p);
    *a11p += tmp3*(*ixx2p)*(*ixx2p) + tmp4*(*ixy2p)*(*ixy2p);
    *a12p += tmp3*(*ixx2p)*(*ixy2p) + tmp4*(*ixy2p)*(*iyy2p);
    *a22p += tmp4*(*iyy2p)*(*iyy2p) + tmp3*(*ixy2p)*(*ixy2p);
    *b1p -=  tmp3*(*ixx2p)*(*ixz2p) + tmp4*(*ixy2p)*(*iyz2p);
    *b2p -=  tmp4*(*iyy2p)*(*iyz2p) + tmp3*(*ixy2p)*(*ixz2p);
    *a11p += tmp5*(*ixx3p)*(*ixx3p) + tmp6*(*ixy3p)*(*ixy3p);
    *a12p += tmp5*(*ixx3p)*(*ixy3p) + tmp6*(*ixy3p)*(*iyy3p);
    *a22p += tmp6*(*iyy3p)*(*iyy3p) + tmp5*(*ixy3p)*(*ixy3p);
    *b1p -=  tmp5*(*ixx3p)*(*ixz3p) + tmp6*(*ixy3p)*(*iyz3p);
    *b2p -=  tmp6*(*iyy3p)*(*iyz3p) + tmp5*(*ixy3p)*(*ixz3p);  
  }

}

__global__ void kernelSubLaplacianVert(
    float *src, float *nextSrc,
    float *dst, float *nextDst,
    float *weights, int height, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tidx < stride) {
    float *wvp    = weights + tidx,
          *srcp   = src + tidx,
          *srcp_s = nextSrc + tidx,
          *dstp   = dst + tidx,
          *dstp_s = nextDst + tidx;

    for (int j = 0; j < height - 1; j++) {
      float tmp = (*wvp) * ((*srcp_s)-(*srcp));
      *dstp += tmp;
      *dstp_s -= tmp;
      wvp += stride; srcp += stride; srcp_s += stride; dstp += stride; dstp_s += stride;
    }
  }

}

__global__ void kernelSubLaplacianHoriz(
    float *src, float *dst, float *weights, float *coeffs, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int col  = tidx % width;

  const int BLOCK_HEIGHT = 1;

  if (tidx < width) {
    float *pSrc         = src + tidx,
          *pDst         = dst + tidx,
          *pWeight      = weights + tidx,
          *pCoeffCalc   = coeffs + tidx,
          *pCoeffUpdate = pCoeffCalc;

    int nBlocks = (height + BLOCK_HEIGHT - 1) / BLOCK_HEIGHT;
    int jCalc = 0;
    int jUpdate = 0;

    // Block calculation and update so coeffs fit in cache

    for (int iBlock = 0; iBlock < nBlocks; iBlock++) {

      // Calc coeffs
      for (int j = 0; j < BLOCK_HEIGHT && jCalc < height; j++, jCalc++) {
        // Do not calculate the last column
        if (col != width - 1)
          *pCoeffCalc = (*pWeight) * ( *(pSrc + 1) - *pSrc );

        pSrc += stride; pWeight += stride; pCoeffCalc += stride;
      }

      // Update dst
      for (int j = 0; j < BLOCK_HEIGHT && jUpdate < height; j++, jUpdate++) {
        float update = 0.0;

        if (col != 0)
          update -= *(pCoeffUpdate - 1);
        if (col != width - 1)
          update += *pCoeffUpdate;

        *pDst += update;

        pDst += stride; pCoeffUpdate += stride;
      }
    }
  }
}

__global__ void kernelSubLaplacianHorizFillCoeffs(
    float *src, float *weights, float *coeffs, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int row  = tidx / stride;
  int col  = tidx % stride;

  // Do not calculate the last column
  if (tidx < width && col != width - 1) {
    float *pSrc    = src + tidx,
          *pWeight = weights + tidx,
          *pCoeff  = coeffs + tidx;

    for (int j = 0; j < height; j++) {
      *pCoeff = (*pWeight) * ( *(pSrc + 1) - *pSrc );

      pSrc += stride; pWeight += stride; pCoeff += stride;
    }
  }

  // // Do not calculate the last column
  // if (col < width - 1) {
  //   float *pSrc    = src + tidx,
  //         *pWeight = weights + tidx,
  //         *pCoeff  = coeffs + tidx;

  //   *pCoeff = (*pWeight) * ( *(pSrc + 1) - *pSrc );
  // }
}

__global__ void kernelSubLaplacianHorizApplyCoeffs(
    float *dst, float *coeffs, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int row  = tidx / stride;
  int col  = tidx % stride;

  if (tidx < width) {

    float *pDst   = dst + tidx,
          *pCoeff = coeffs + tidx;

    for (int j = 0; j < height; j++) {
      float update = 0.0;

      if (col != 0)
        update -= *(pCoeff - 1);
      if (col != width - 1)
        update += *pCoeff;

      *pDst += update;

      pDst += stride; pCoeff += stride;
    }
  }

  // if (col < width) {

  //   float *pDst   = dst + tidx,
  //         *pCoeff = coeffs + tidx;

  //   float update = 0.0;

  //   if (col != 0)
  //     update -= *(pCoeff - 1);
  //   if (col != width - 1)
  //     update += *pCoeff;

  //   *pDst += update;
  // }
}

__global__ void kernelSorStep(
    float *du, float *dv,
    float *a11, float *a12, float *a22,
    const float *b1, const float *b2,
    const float *horiz, const float *vert,
    const int iterations, const float omega,
    int height, int width, int stride, bool odd) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int j  = tidx / width;
  int i  = tidx % width;

  bool shouldRun = (odd)
    ? ((i + j) % 2 == 1)
    : ((i + j) % 2 == 0);

  if (tidx < width * height && shouldRun) {

    float sigma_u,sigma_v,sum_dpsis,A11,A22,A12,B1,B2;
    sigma_u = 0.0f;
    sigma_v = 0.0f;
    sum_dpsis = 0.0f;

    int here  = j * stride + i;
    int left  = j * stride + i - 1;
    int right = j * stride + i + 1;
    int up    = (j-1) * stride + i;
    int down  = (j+1) * stride + i;

    if(j>0)
    {
      sigma_u   -= vert[up] * du[up];
      sigma_v   -= vert[up] * dv[up];
      sum_dpsis += vert[up];
    }
    if(i>0)
    {
      sigma_u   -= horiz[left] * du[left];
      sigma_v   -= horiz[left] * dv[left];
      sum_dpsis += horiz[left];
    }
    if(j<height-1)
    {
      sigma_u   -= vert[here] * du[down];
      sigma_v   -= vert[here] * dv[down];
      sum_dpsis += vert[here];
    }
    if(i<width-1)
    {
      sigma_u   -= horiz[here] * du[right];
      sigma_v   -= horiz[here] * dv[right];
      sum_dpsis += horiz[here];
    }

    A11 = a11[here] + sum_dpsis;
    A12 = a12[here];
    A22 = a22[here] + sum_dpsis;

    B1 = b1[here] - sigma_u;
    B2 = b2[here] - sigma_v;

    du[here] = (1.0f-omega) * du[here] + omega/A11 * (B1 - A12 * dv[here]);
    dv[here] = (1.0f-omega) * dv[here] + omega/A22 * (B2 - A12 * du[here]);

  }
}

__global__ void kernelGetMeanImageAndDiff(
    float *img1, float *img2, float *avgImg, float *diff,
    int height, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  // For 3 channels images
  if (tidx < 3 * stride) {
    float
      *pImg1   = img1 + tidx,
      *pImg2   = img2 + tidx,
      *pAvgImg = avgImg + tidx,
      *pDiff   = diff + tidx;

    for (int j = 0; j < height; j++) {
      *pAvgImg = 0.5 * ((*pImg1) + (*pImg2));
      *pDiff   = (*pImg2) - (*pImg1);

      pImg1 += stride; pImg2 += stride; pAvgImg += stride; pDiff += stride;
    }

  }

}


__global__ void kernelFlowMag(
    float *dst,  float *ux,  float *uy,  float *vx,  float *vy,
    float qa, float epsmooth, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tidx < height * stride) {
    float *uxp = ux + tidx,
          *uyp = uy + tidx,
          *vxp = vx + tidx,
          *vyp = vy + tidx,
          *sp  = dst + tidx;

    *sp = qa / sqrtf(
        (*uxp)*(*uxp) + (*uyp)*(*uyp) + (*vxp)*(*vxp) + (*vyp)*(*vyp) + epsmooth );

  }
}

__global__ void kernelSmoothnessHorizVert(
    float *dst_horiz, float *dst_vert, float *smoothness, int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tidx < height * stride) {

    float *dst_horiz_p = dst_horiz + tidx,
          *dst_vert_p  = dst_vert  + tidx,
          *sp          = smoothness + tidx;

    *dst_horiz_p = *sp + *(sp + 1);
    *dst_vert_p  = *sp + *(sp + stride);
  }
}


__global__ void kernelFlowUpdate(
    float *uu, float *vv, float *wx, float *wy, float *du, float *dv,
    int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tidx < height * stride) {

    float *uup = uu + tidx,
          *vvp = vv + tidx,
          *wxp = wx + tidx,
          *wyp = wy + tidx,
          *dup = du + tidx,
          *dvp = dv + tidx;

    (*uup) = (*wxp) + (*dup);
    (*vvp) = (*wyp) + (*dvp);
  }

}


__global__ void kernelWarpImage(
    float *dst1, float *dst2, float *dst3, float *mask,
    float *src1, float *src2, float *src3,
    float *wx, float *wy,
    int height, int width, int stride) {

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  int i = tidx % stride;
  int j = tidx / stride;
  int offset = j * stride + i;

  if (i < width && j < height) {
    float xx = i + wx[offset];
    float yy = j + wy[offset];
    int x = floor(xx);
    int y = floor(yy);
    float dx = xx - x;
    float dy = yy - y;

    // Set mask according to bounds
    mask[offset] = (xx >= 0 && xx < width && yy >= 0 && yy < height);

    int x1 = MINMAX_TA(x, width);
    int x2 = MINMAX_TA(x + 1, width);
    int y1 = MINMAX_TA(y, height);
    int y2 = MINMAX_TA(y + 1, height);

    dst1[offset] = 
      src1[y1 * stride + x1] * (1.0f-dx) * (1.0f-dy) +
      src1[y1 * stride + x2] * dx * (1.0f-dy) +
      src1[y2 * stride + x1] * (1.0f-dx) * dy +
      src1[y2 * stride + x2] * dx * dy;
    dst2[offset] = 
      src2[y1 * stride + x1] * (1.0f-dx) * (1.0f-dy) +
      src2[y1 * stride + x2] * dx * (1.0f-dy) +
      src2[y2 * stride + x1] * (1.0f-dx) * dy +
      src2[y2 * stride + x2] * dx * dy;
    dst3[offset] = 
      src3[y1 * stride + x1] * (1.0f-dx) * (1.0f-dy) +
      src3[y1 * stride + x2] * dx * (1.0f-dy) +
      src3[y2 * stride + x1] * (1.0f-dx) * dy +
      src3[y2 * stride + x2] * dx * dy;

  }
}


namespace cu {

  void dataTerm(
      image_t *a11, image_t *a12, image_t *a22,
      image_t *b1, image_t *b2, 
      image_t *mask, 
      image_t *wx, image_t *wy,
      image_t *du, image_t *dv, 
      image_t *uu, image_t *vv, 
      color_image_t *Ix,  color_image_t *Iy,  color_image_t *Iz,
      color_image_t *Ixx, color_image_t *Ixy, color_image_t *Iyy,
      color_image_t *Ixz, color_image_t *Iyz, 
      const float half_delta_over3, const float half_beta, const float half_gamma_over3) {

    memset(a11->c1, 0, sizeof(float)*uu->height*uu->stride);
    memset(a12->c1, 0, sizeof(float)*uu->height*uu->stride);
    memset(a22->c1, 0, sizeof(float)*uu->height*uu->stride);
    memset(b1->c1 , 0, sizeof(float)*uu->height*uu->stride);
    memset(b2->c1 , 0, sizeof(float)*uu->height*uu->stride);

    // Set up device pointers
    float *a11c1,
          *a12c1,    *a22c1,
          *b1c1,     *b2c1, 
          *maskc1, 
          *wxc1,     *wyc1,
          *duc1,     *dvc1, 
          *uuc1,     *vvc1, 
          *Ixc1,     *Ixc2,     *Ixc3,
          *Iyc1,     *Iyc2,     *Iyc3,
          *Izc1,     *Izc2,     *Izc3,
          *Ixxc1,    *Ixxc2,    *Ixxc3,
          *Ixyc1,    *Ixyc2,    *Ixyc3,
          *Iyyc1,    *Iyyc2,    *Iyyc3,
          *Ixzc1,    *Ixzc2,    *Ixzc3,
          *Iyzc1,    *Iyzc2,    *Iyzc3;

    checkCudaErrors( hipHostGetDevicePointer(&a11c1,    a11->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&a12c1,    a12->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&a22c1,    a22->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&b1c1,     b1->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&b2c1,     b2->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&maskc1,   mask->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&wxc1,     wx->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&wyc1,     wy->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&duc1,     du->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&dvc1,     dv->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&uuc1,     uu->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&vvc1,     vv->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixc1,     Ix->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixc2,     Ix->c2, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixc3,     Ix->c3, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Iyc1,     Iy->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Iyc2,     Iy->c2, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Iyc3,     Iy->c3, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Izc1,     Iz->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Izc2,     Iz->c2, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Izc3,     Iz->c3, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixxc1,    Ixx->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixxc2,    Ixx->c2, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixxc3,    Ixx->c3, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixyc1,    Ixy->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixyc2,    Ixy->c2, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixyc3,    Ixy->c3, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Iyyc1,    Iyy->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Iyyc2,    Iyy->c2, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Iyyc3,    Iyy->c3, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixzc1,    Ixz->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixzc2,    Ixz->c2, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Ixzc3,    Ixz->c3, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Iyzc1,    Iyz->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Iyzc2,    Iyz->c2, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&Iyzc3,    Iyz->c3, 0) );

    int N = uu->height*uu->stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelDataTerm<<<nBlocks, nThreadsPerBlock>>>(
        a11c1, a12c1, a22c1,
        b1c1, b2c1, 
        maskc1, 
        wxc1, wyc1,
        duc1, dvc1, 
        uuc1, vvc1, 
        Ixc1,    Ixc2,    Ixc3,
        Iyc1,    Iyc2,    Iyc3,
        Izc1,    Izc2,    Izc3,
        Ixxc1,   Ixxc2,   Ixxc3,
        Ixyc1,   Ixyc2,   Ixyc3,
        Iyyc1,   Iyyc2,   Iyyc3,
        Ixzc1,   Ixzc2,   Ixzc3,
        Iyzc1,   Iyzc2,   Iyzc3, 
        half_delta_over3, half_beta, half_gamma_over3, N);

  };


  void subLaplacianHoriz(
      float *src, float *dst, float *weights, int height, int width, int stride) {

    float *pDeviceCoeffs;
    checkCudaErrors( hipMalloc((void**) &pDeviceCoeffs, height * stride * sizeof(float)) );

    // Setup device pointers
    float *pDeviceSrc, *pDeviceDst, *pDeviceWeights;
    checkCudaErrors( hipHostGetDevicePointer(&pDeviceSrc, src, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&pDeviceDst, dst, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&pDeviceWeights, weights, 0) );

    int N = width;
    // int N = height * stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    auto start_horiz = now();

    kernelSubLaplacianHorizFillCoeffs<<<nBlocks, nThreadsPerBlock>>>(
        pDeviceSrc, pDeviceWeights, pDeviceCoeffs, height, width, stride);

    kernelSubLaplacianHorizApplyCoeffs<<<nBlocks, nThreadsPerBlock>>>(
        pDeviceDst, pDeviceCoeffs, height, width, stride);

    // kernelSubLaplacianHoriz<<<nBlocks, nThreadsPerBlock>>>(
    //     pDeviceSrc, pDeviceDst, pDeviceWeights, pDeviceCoeffs, height, width, stride);
    hipDeviceSynchronize();
    // calc_print_elapsed("laplacian horiz", start_horiz);

    hipFree(pDeviceCoeffs);
  }

  void subLaplacianVert(
      float *src, float *dst, float *weights, int height, int stride) {

    float *d_src, *d_dst, *d_weights;

    checkCudaErrors( hipHostGetDevicePointer(&d_src, src, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_dst, dst, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_weights, weights, 0) );

    int N = stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    auto start_vert = now();
    kernelSubLaplacianVert<<<nBlocks, nThreadsPerBlock>>>(
        d_src, d_src + stride, d_dst, d_dst + stride, d_weights, height, stride);
    // calc_print_elapsed("laplacian vert", start_vert);

  }

  void sor(
      float *du, float *dv,
      float *a11, float *a12, float *a22,
      float *b1, float *b2,
      float *horiz, float *vert,
      int iterations, float omega,
      int height, int width, int stride) {

    // Device setup
    float 
      *d_du,
    *d_dv,
    *d_a11,
    *d_a12,
    *d_a22,
    *d_b1,
    *d_b2,
    *d_horiz,
    *d_vert;

    checkCudaErrors( hipHostGetDevicePointer(&d_du,    du, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_dv,    dv, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_a11,   a11, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_a12,   a12, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_a22,   a22, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_b1,    b1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_b2,    b2, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_horiz, horiz, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_vert,  vert, 0) );

    int N = width * height;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    for(int iter = 0 ; iter<iterations ; iter++)
    {

      kernelSorStep<<<nBlocks, nThreadsPerBlock>>>(
          d_du, d_dv,
          d_a11, d_a12, d_a22,
          d_b1, d_b2,
          d_horiz, d_vert,
          iterations, omega,
          height, width, stride, true);

      hipDeviceSynchronize();

      kernelSorStep<<<nBlocks, nThreadsPerBlock>>>(
          d_du, d_dv,
          d_a11, d_a12, d_a22,
          d_b1, d_b2,
          d_horiz, d_vert,
          iterations, omega,
          height, width, stride, false);
    }  
  }

  void getMeanImageAndDiff(
      float *img1, float *img2, float *avgImg, float *diff,
      int height, int stride) {

    float
      *d_img1,
    *d_img2,
    *d_avgImg,
    *d_diff;

    checkCudaErrors( hipHostGetDevicePointer(&d_img1,   img1,   0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_img2,   img2,   0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_avgImg, avgImg, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_diff,   diff,   0) );

    int N = 3 * stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelGetMeanImageAndDiff<<<nBlocks, nThreadsPerBlock>>>(
        d_img1, d_img2, d_avgImg, d_diff,
        height, stride);

  }

  void colorImageDerivative(
      float *dst, float *src, float *pDeviceColorDerivativeKernel, 
      int height, int width, int stride, bool horiz) {

    float *d_dst, *d_src;

    checkCudaErrors( hipHostGetDevicePointer(&d_dst, dst, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_src, src, 0) );

    Npp32f *pDeviceSrc = d_src;
    Npp32f *pDeviceDst = d_dst;

    size_t elemSize = sizeof(float);
    unsigned int nSrcStep = stride * elemSize;
    unsigned int nDstStep = nSrcStep;

    NppiSize oSrcSize = { width, height };
    NppiPoint oSrcOffset = { 0, 0 };
    NppiSize oSizeROI = { width, height };
    NppiBorderType eBorderType = NPP_BORDER_REPLICATE;

    NPP_CHECK_NPP(
        (horiz)
        ? nppiFilterRowBorder_32f_C1R (
          pDeviceSrc, nSrcStep, oSrcSize, oSrcOffset,
          pDeviceDst, nDstStep, oSizeROI,
          pDeviceColorDerivativeKernel, 5, 2, eBorderType)
        : nppiFilterColumnBorder_32f_C1R (
          pDeviceSrc, nSrcStep, oSrcSize, oSrcOffset,
          pDeviceDst, nDstStep, oSizeROI,
          pDeviceColorDerivativeKernel, 5, 2, eBorderType)
        );
  }

  // Expects filter kernel of the form
  //   { -0.5, 0.0, 0.5 }
  void imageDerivative(
      float *dst, float *src, float *pDeviceDerivativeKernel, 
      int height, int width, int stride, bool horiz) {

    float *d_dst, *d_src;

    checkCudaErrors( hipHostGetDevicePointer(&d_dst, dst, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_src, src, 0) );

    Npp32f *pDeviceSrc = d_src;
    Npp32f *pDeviceDst = d_dst;

    size_t elemSize = sizeof(float);
    unsigned int nSrcStep = stride * elemSize;
    unsigned int nDstStep = nSrcStep;

    NppiSize oSrcSize = { width, height };
    NppiPoint oSrcOffset = { 0, 0 };
    NppiSize oSizeROI = { width, height };
    NppiBorderType eBorderType = NPP_BORDER_REPLICATE;

    NPP_CHECK_NPP(
        (horiz)
        ? nppiFilterRowBorder_32f_C1R (
          pDeviceSrc, nSrcStep, oSrcSize, oSrcOffset,
          pDeviceDst, nDstStep, oSizeROI,
          pDeviceDerivativeKernel, 3, 1, eBorderType)
        : nppiFilterColumnBorder_32f_C1R (
          pDeviceSrc, nSrcStep, oSrcSize, oSrcOffset,
          pDeviceDst, nDstStep, oSizeROI,
          pDeviceDerivativeKernel, 3, 1, eBorderType)
        );
  }

  void smoothnessTerm(
      float *dst_horiz, float *dst_vert, float *smoothness,
      float *ux,  float *uy,  float *vx,  float *vy,
      float qa, float epsmooth,
      int height, int width, int stride) {

    float *d_dst_horiz, *d_dst_vert, *d_smoothness, *d_ux, *d_uy, *d_vx, *d_vy;

    checkCudaErrors( hipHostGetDevicePointer(&d_dst_horiz,     dst_horiz,  0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_dst_vert,      dst_vert,   0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_smoothness,    smoothness, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_ux,            ux,         0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_uy,            uy,         0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_vx,            vx,         0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_vy,            vy,         0) );

    int N = height * stride;
    int nThreadsPerBlock = 128;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelFlowMag<<<nBlocks, nThreadsPerBlock>>> (
        d_smoothness, d_ux, d_uy, d_vx, d_vy,
        qa, epsmooth, height, width, stride);

    hipDeviceSynchronize();

    kernelSmoothnessHorizVert<<< nBlocks, nThreadsPerBlock >>> (
        d_dst_horiz, d_dst_vert, d_smoothness, height, width, stride);

    hipDeviceSynchronize();
  }


  void flowUpdate(
      float *uu, float *vv, float *wx, float *wy, float *du, float *dv,
      int height, int width, int stride) {

    float *d_uu,
          *d_vv,
          *d_wx,
          *d_wy,
          *d_du,
          *d_dv;

    checkCudaErrors( hipHostGetDevicePointer(&d_uu, uu, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_vv, vv, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_wx, wx, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_wy, wy, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_du, du, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_dv, dv, 0) );

    int N = height * stride;
    int nThreadsPerBlock = 128;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelFlowUpdate<<< nBlocks, nThreadsPerBlock >>> (
        d_uu, d_vv, d_wx, d_wy, d_du, d_dv,
        height, width, stride);

  }

  /*
     Warp an image `src` into `dst` using warp vectors `wx`, `wy`.
     Store `mask[i]` = 0 or 1 if pixel i goes outisde or inside image bounds.
   */
  void warpImage(
      color_image_t *dst, image_t *mask, const color_image_t *src, const image_t *wx, const image_t *wy) {

    float *d_dst1, *d_dst2, *d_dst3;
    float *d_src1, *d_src2, *d_src3;
    float *d_mask, *d_wx,   *d_wy;

    checkCudaErrors( hipHostGetDevicePointer(&d_dst1, dst->c1,  0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_dst2, dst->c2,  0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_dst3, dst->c3,  0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_src1, src->c1,  0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_src2, src->c2,  0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_src3, src->c3,  0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_mask, mask->c1, 0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_wx,   wx->c1,   0) );
    checkCudaErrors( hipHostGetDevicePointer(&d_wy,   wy->c1,   0) );

    int N = src->height * src->stride;
    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelWarpImage<<< nBlocks, nThreadsPerBlock >>> (
        d_dst1, d_dst2, d_dst3, d_mask,
        d_src1, d_src2, d_src3,
        d_wx,   d_wy, src->height, src->width, src->stride);
  }

}
