#include "hip/hip_runtime.h"
/**
 * Implements kernels for flow densification
 */

// System
#include <iostream>
#include <chrono>
#include <string>
#include <stdexcept>

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "densify.h"

__global__ void kernelDensifyPatch(
    float* pDeviceCostDiff, float* pDeviceFlowOut, float* pDeviceWeights,
    dev_patch_state* states, int ip,
    int midpointX, int midpointY,
    int width, int height,
    int patchSize, float minErrVal) {

  float flowX = states[ip].p_curx;
  float flowY = states[ip].p_cury;
  int lower_bound = -patchSize / 2;

  int x = threadIdx.x + lower_bound;
  int y = blockIdx.x  + lower_bound;

  int xt = x + midpointX;
  int yt = y + midpointY;

  if (xt >= 0 && yt >= 0 && xt < width && yt < height) {

    int i = yt * width + xt;
    int j = blockIdx.x * patchSize + threadIdx.x;

    float absw = (float) (fmaxf(minErrVal, pDeviceCostDiff[3 * j]));
    absw += (float) (fmaxf(minErrVal, pDeviceCostDiff[3 * j + 1]));
    absw += (float) (fmaxf(minErrVal, pDeviceCostDiff[3 * j + 2]));
    absw = 1.0 / absw;

    // Weight contribution RGB
    pDeviceWeights[i] += absw;

    pDeviceFlowOut[2 * i] += flowX * absw;
    pDeviceFlowOut[2 * i + 1] += flowY * absw;
  }

}


__global__ void kernelDensifyPatches(
    float** costs, float* flow, float* weights,
    float* flowXs, float* flowYs, bool* valid,
    float* midpointX, float* midpointY,
    int width, int height,
    int patch_size, float minErrVal) {

  int patchId = blockIdx.x;
  int tid = threadIdx.x;
  if (!valid[patchId]) return;

  int lower_bound = -patch_size / 2;
  int xt = midpointX[patchId] + lower_bound;
  int yt = midpointY[patchId] + lower_bound;
  int offset = (xt + yt * width) + tid;

  float* cost = costs[patchId];

  for (int i = 3 * tid, j = offset; i < patch_size * patch_size * 3;
      i += 3 * patch_size, j += width) {

    if (j >= 0 && j < width * height) {

      float absw = (float) (fmaxf(minErrVal, cost[i]));
      absw += (float) (fmaxf(minErrVal, cost[i + 1]));
      absw += (float) (fmaxf(minErrVal, cost[i + 2]));
      absw = 1.0 / absw;

      // Weight contribution RGB
      atomicAdd(&weights[j], absw);

      atomicAdd(&flow[2 * j], flowXs[patchId] * absw);
      atomicAdd(&flow[2 * j + 1], flowYs[patchId] * absw);
    }

  }

}


__global__ void kernelNormalizeFlow(
    float* pDeviceFlowOut, float* pDeviceWeights, int N) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N && pDeviceWeights[i] > 0) {
    pDeviceFlowOut[2 * i]     /= pDeviceWeights[i];
    pDeviceFlowOut[2 * i + 1] /= pDeviceWeights[i];
  }

}

namespace cu {

  void densifyPatch(
      float* pDeviceCostDiff, float* pDeviceFlowOut, float* pDeviceWeights,
      dev_patch_state* states, int ip,
      int midpointX, int midpointY,
      int width, int height,
      int patchSize, float minErrVal) {

    int nBlocks = patchSize;
    int nThreadsPerBlock = patchSize;

    kernelDensifyPatch<<<nBlocks, nThreadsPerBlock>>>(
        pDeviceCostDiff, pDeviceFlowOut, pDeviceWeights,
        states, ip,
        midpointX, midpointY,
        width, height,
        patchSize, minErrVal);
  }

  void normalizeFlow(
      float* pDeviceFlowOut, float* pDeviceWeights, int N) {

    int nThreadsPerBlock = 64;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;

    kernelNormalizeFlow<<<nBlocks, nThreadsPerBlock>>>(pDeviceFlowOut, pDeviceWeights, N);
  }

  void densifyPatches(
      float** costs, float* flow, float* weights,
      float* flowXs, float* flowYs, bool* valid,
      float* midpointX, float* midpointY, int n_patches,
      const opt_params* op, const img_params* i_params) {

    int nBlocks = n_patches;
    int nThreadsPerBlock = op->patch_size;

    kernelDensifyPatches<<<nBlocks, nThreadsPerBlock>>>(
        costs, flow, weights,
        flowXs, flowYs, valid,
        midpointX,  midpointY,
        i_params->width, i_params->height,
        op->patch_size, op->min_errval);

  }

}
