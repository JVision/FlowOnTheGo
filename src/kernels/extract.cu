#include "hip/hip_runtime.h"
/**
 * Implements a patch extraction kernel
 */

// System
#include <iostream>
#include <chrono>
#include <string>
#include <stdexcept>

// CUDA
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../common/cuda_helper.h"
#include "../common/Exceptions.h"
#include "../common/timer.h"


#include "extract.h"

__global__  void kernelExtractPatch(
    float* pDevicePatch, float* pDevicePatchX, float* pDevicePatchY,
    const float* I0, const float* I0x, const float* I0y, int patch_offset,
    int patchSize, int width_pad) {

  int patchIdx = threadIdx.x + blockIdx.x * 3 * patchSize;
  int imgIdx = 3 * patch_offset + threadIdx.x + blockIdx.x * 3 * width_pad;

  pDevicePatch[patchIdx] = I0[imgIdx];
  pDevicePatchX[patchIdx] = I0x[imgIdx];
  pDevicePatchY[patchIdx] = I0y[imgIdx];

}


__global__ void kernelExtractPatchesAndHessians(
    float** patches, float** patchxs, float** patchys,
    const float * I0, const float * I0x, const float * I0y,
    float* H00, float* H01, float* H11,
    float** tempXX, float** tempXY, float** tempYY,
    float* midpointX, float* midpointY, int padding,
    int patch_size, int width_pad) {


  int patchId = blockIdx.x;
  int tid = threadIdx.x;
  float* patch = patches[patchId];
  float* patchX = patchxs[patchId];
  float* patchY = patchys[patchId];
  float* XX = tempXX[patchId];
  float* XY = tempXY[patchId];
  float* YY = tempYY[patchId];

  int x = round(midpointX[patchId]) + padding;
  int y = round(midpointY[patchId]) + padding;

  int lb = -patch_size / 2;
  int offset = 3 * ((x + lb) + (y + lb) * width_pad) + tid;

  for (int i = tid, j = offset; i < patch_size * patch_size * 3;
      i += 3 * patch_size, j += 3 * width_pad) {
    patch[i] = I0[j];
    patchX[i] = I0x[j];
    patchY[i] = I0y[j];
    XX[i] = patchX[i] * patchX[i];
    XY[i] = patchX[i] * patchY[i];
    YY[i] = patchY[i] * patchY[i];
  }

  __syncthreads();

  // Mean normalize
  __shared__ float mean;

  if (tid == 0) {

    mean = 0.0;
    for (int i = 0; i < patch_size * patch_size * 3; i++) {
      mean += patch[i];
    }
    mean /= patch_size * patch_size * 3;

  }

  __syncthreads();

  for (int i = tid; i < patch_size * patch_size * 3;
      i+= 3 * patch_size) {
    patch[i] -= mean;
  }

  // TODO: can this be done in parallel?
  if (tid == 0) {

    float h00 = 0.0, h01 = 0.0, h11 = 0.0;

    for (int i = 0; i < patch_size * patch_size * 3; i++) {
      h00 += XX[i];
      h01 += XY[i];
      h11 += YY[i];
    }

    // If not invertible adjust values
    if (h00 * h11 - h01 * h01 == 0) {
      h00 += 1e-10;
      h11 += 1e-10;
    }

    H00[patchId] = h00;
    H01[patchId] = h01;
    H11[patchId] = h11;

  }


}


namespace cu {

  void extractPatch(
      float* pDevicePatch, float* pDevicePatchX, float* pDevicePatchY,
      const float* I0, const float* I0x, const float* I0y, int patch_offset,
      int patchSize, int width_pad) {

    int nBlocks = patchSize;
    int nThreadsPerBlock = 3 * patchSize;

    kernelExtractPatch<<<nBlocks, nThreadsPerBlock>>>(
        pDevicePatch, pDevicePatchX, pDevicePatchY,
        I0, I0x, I0y, patch_offset,
        patchSize, width_pad);

  }


  void extractPatchesAndHessians(
      float** patches, float** patchxs, float** patchys,
      const float * I0, const float * I0x, const float * I0y,
      float* H00, float* H01, float* H11,
      float** tempXX, float** tempXY, float** tempYY,
      float* midpointX, float* midpointY, int n_patches,
      const opt_params* op, const img_params* i_params) {

    int nBlocks = n_patches;
    int nThreadsPerBlock = 3 * op->patch_size;

    kernelExtractPatchesAndHessians<<<nBlocks, nThreadsPerBlock>>>(
        patches, patchxs, patchys,
        I0, I0x, I0y, H00, H01, H11,
        tempXX, tempXY, tempYY, midpointX, midpointY,
        i_params->padding, op->patch_size, i_params->width_pad);

  }

}
