#include "hip/hip_runtime.h"
/**
 * Implements kernels for optimization steps
 * Mainly interpolating patch, computing cost error,
 * calculating deltap and computing cost error.
 */

// System
#include <iostream>
#include <chrono>
#include <string>
#include <stdexcept>

// CUDA
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../common/cuda_helper.h"
#include "../common/Exceptions.h"
#include "../common/timer.h"


#include "optimize.h"

__device__ void calcProjection(dev_patch_state* states,
    float* patchX, float* patchY, float* tempX, float* tempY,
    float* raw, int patch_size, int out_thresh,
    int lb, int ubw, int ubh) {

  int patchId = blockIdx.x;
  int tid = threadIdx.x;

  for (int i = tid; i < 3 * patch_size * patch_size; i += 3 * patch_size) {
    tempX[i] = patchX[i] * raw[i];
    tempY[i] = patchY[i] * raw[i];
  }

  __syncthreads();

  if (tid == 0) {
    states[patchId].count++;

    float dpx = 0.0;
    float dpy = 0.0;
    for (int i = 0; i < 3 * patch_size * patch_size; i++) {
      dpx += tempX[i];
      dpy += tempY[i];
    }

    float det = states[patchId].H00 * states[patchId].H11
      - states[patchId].H01 * states[patchId].H01;
    states[patchId].delta_px = states[patchId].H11 * dpx - states[patchId].H01 * dpy;
    states[patchId].delta_py = states[patchId].H00 * dpy - states[patchId].H01 * dpx;
    states[patchId].delta_px /= det;
    states[patchId].delta_py /= det;

    // Update flow
    states[patchId].p_curx -= states[patchId].delta_px;
    states[patchId].p_cury -= states[patchId].delta_py;

    // Update midpoint
    states[patchId].midpoint_curx = states[patchId].midpoint_orgx 
      + states[patchId].p_curx;
    states[patchId].midpoint_cury = states[patchId].midpoint_orgy 
      + states[patchId].p_cury;

    // Outlier check
    float norm = sqrt((states[patchId].midpoint_curx - states[patchId].midpoint_orgx) 
        * (states[patchId].midpoint_curx - states[patchId].midpoint_orgx)
        + (states[patchId].midpoint_cury - states[patchId].midpoint_orgy)
        * (states[patchId].midpoint_cury - states[patchId].midpoint_orgy));

    if (norm > out_thresh || states[patchId].midpoint_curx < lb 
        || states[patchId].midpoint_cury < lb
        || states[patchId].midpoint_curx > ubw 
        || states[patchId].midpoint_cury > ubh) {

      states[patchId].p_curx = states[patchId].p_orgx;
      states[patchId].p_cury = states[patchId].p_orgy;

      // Update midpoint
      states[patchId].midpoint_curx = states[patchId].midpoint_orgx 
        + states[patchId].p_curx;
      states[patchId].midpoint_cury = states[patchId].midpoint_orgy 
        + states[patchId].p_cury;

      states[patchId].has_converged = 1;
      states[patchId].has_opt_started = 1;

    }

  }

}


__global__ void kernelInterpolateAndComputeErr(
    dev_patch_state* states, float** raw_diff, float** costs,
    float** patches, float** patchXs, float** patchYs, const float* I1,
    float** tempXX, float** tempYY,
    int n_patches, int padding, int patch_size,
    int width_pad, int gd_iter, float res_thresh, float dp_thresh,
    float dr_thresh, float out_thresh, int lb, int ubw, int ubh) {

  int patchId = blockIdx.x;
  int tid = threadIdx.x;
  float* raw = raw_diff[patchId];
  float* cost = costs[patchId];
  float* patch = patches[patchId];
  float* patchX = patchXs[patchId];
  float* patchY = patchYs[patchId];
  float* tempX = tempXX[patchId];
  float* tempY = tempYY[patchId];
  bool notFirst = false;

  while (!states[patchId].has_converged) {

    if (notFirst) {
      calcProjection(states, patchX, patchY, tempX, tempY, raw, patch_size,
          out_thresh, lb, ubw, ubh);
    }


    // Interpolate the patch

    float pos0, pos1, pos2, pos3, resid0, resid1, w0, w1, w2, w3;

    // Compute the bilinear weight vector, for patch without orientation/scale change
    // weight vector is constant for all pixels
    // TODO: compare performance when shared and only tid 0 does this precomp
    pos0 = ceil(states[patchId].midpoint_curx + .00001f); // ensure rounding up to natural numbers
    pos1 = ceil(states[patchId].midpoint_cury + .00001f);
    pos2 = floor(states[patchId].midpoint_curx);
    pos3 = floor(states[patchId].midpoint_cury);

    resid0 = states[patchId].midpoint_curx - (float)pos2;
    resid1 = states[patchId].midpoint_cury - (float)pos3;
    w0 = resid0 * resid1;
    w1 = (1 - resid0) * resid1;
    w2 = resid0 * (1- resid1);
    w3 = (1 - resid0) * (1 - resid1);

    pos0 += padding;
    pos1 += padding;

    int lb = -patch_size / 2;
    int x = 3 * (pos0 + lb) + tid;
    int starty = pos1 + lb;


    for (int i = tid, j = starty; i < patch_size * patch_size * 3;
        i += 3 * patch_size, j += 3 * width_pad) {

      const float* img_e = I1 + x;
      const float* img_a = img_e + j * width_pad * 3;
      const float* img_c = img_e + (j - 1) * width_pad * 3;
      const float* img_b = img_a - 3;
      const float* img_d = img_c - 3;
      raw[i] = w0 * (*img_a) + w1 * (*img_b) + w2 * (*img_c) + w3 * (*img_d);

    }

    // Compute mean
    __shared__ float mean;

    if (tid == 0) {

      mean = 0.0;
      for (int i = 0; i < patch_size * patch_size * 3; i++) {
        mean += raw[i];
      }
      mean /= patch_size * patch_size * 3;

    }

    __syncthreads();

    // Normalize and compute cost
    for (int i = tid; i < patch_size * patch_size * 3;
        i+= 3 * patch_size) {
      raw[i] -= mean;
      raw[i] -= patch[i];
      cost[i] = raw[i] * raw[i];
    }

    if (tid == 0) {
      float c = 0.0;
      for (int i = 0; i < patch_size * patch_size * 3; i++) {
        c += cost[i];
      }
      states[patchId].cost = c;

      // Check convergence

      // Compute step norm
      states[patchId].delta_p_sq_norm = 
        states[patchId].delta_px * states[patchId].delta_px + 
        states[patchId].delta_py * states[patchId].delta_py;

      if (states[patchId].count == 1)
        states[patchId].delta_p_sq_norm_init = states[patchId].delta_p_sq_norm;

      // Check early termination criterions
      states[patchId].mares_old = states[patchId].mares;
      states[patchId].mares = c / (3 * patch_size * patch_size);

      if (!((states[patchId].count < gd_iter) & (states[patchId].mares > res_thresh)
            & ((states[patchId].count < gd_iter) 
              | (states[patchId].delta_p_sq_norm / states[patchId].delta_p_sq_norm_init >= dp_thresh))
            & ((states[patchId].count < gd_iter) 
              | (states[patchId].mares / states[patchId].mares_old <= dr_thresh)))) {

        states[patchId].has_converged = 1;

      }

    }

    notFirst = true;

  }

}



namespace cu {

  void interpolateAndComputeErr(dev_patch_state* states,
      float** raw_diff, float** costs, float** patches, float** patchXs,
      float** patchYs, float** tempXX, float** tempYY, const float* I1,
      int n_patches, const opt_params* op,
      const img_params* i_params) {

    int nBlocks = n_patches;
    int nThreadsPerBlock = 3 * op->patch_size;

    kernelInterpolateAndComputeErr<<<nBlocks, nThreadsPerBlock>>>(
        states, raw_diff, costs, patches, patchXs, patchYs, I1,
        tempXX, tempYY, n_patches,
        i_params->padding, op->patch_size, i_params->width_pad, 
        op->grad_descent_iter, op->res_thresh, op->dp_thresh,
        op->dr_thresh, op->outlier_thresh, i_params->l_bound, i_params->u_bound_width,
        i_params->u_bound_height);


  }

}
