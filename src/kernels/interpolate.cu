#include "hip/hip_runtime.h"
/**
 * Implements a bilinear interpolation kernel
 */

// System
#include <iostream>
#include <chrono>
#include <string>
#include <stdexcept>

// CUDA
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../common/cuda_helper.h"
#include "../common/Exceptions.h"
#include "../common/timer.h"


#include "interpolate.h"

__global__ void kernelInterpolatePatch(
    float* pDeviceRawDiff, const float* pDeviceI, float* weight,
    int width_pad, int starty, int startx, int patchSize) {

  int x = threadIdx.x + startx;
  int y = blockIdx.x  + starty;
  int patchIdx = threadIdx.x + blockIdx.x * patchSize;

  if (x < startx + patchSize && y < starty + patchSize) {
    const float* img_e = pDeviceI + x * 3;
    const float* img_a = img_e + y * width_pad * 3;
    const float* img_c = img_e + (y - 1) * width_pad * 3;
    const float* img_b = img_a - 3;
    const float* img_d = img_c - 3;

    int diff = x * 3 + y * width_pad * 3;
    pDeviceRawDiff[3 * patchIdx] =
      weight[0] * (*img_a) + weight[1] * (*img_b) + weight[2] * (*img_c) + weight[3] * (*img_d);
    ++img_a; ++img_b; ++img_c; ++img_d;
    pDeviceRawDiff[3 * patchIdx + 1] =
      weight[0] * (*img_a) + weight[1] * (*img_b) + weight[2] * (*img_c) + weight[3] * (*img_d);
    ++img_a; ++img_b; ++img_c; ++img_d;
    pDeviceRawDiff[3 * patchIdx + 2] =
      weight[0] * (*img_a) + weight[1] * (*img_b) + weight[2] * (*img_c) + weight[3] * (*img_d);
  }

}

__global__ void kernelNormalizeMean(
    float* src, float mean, int patch_size) {

  int i = blockIdx.x * patch_size + threadIdx.x;

  src[3 * i]     -= mean;
  src[3 * i + 1] -= mean;
  src[3 * i + 2] -= mean;

}


namespace cu {

  void interpolatePatch(
      float* pDeviceRawDiff, const float* pDeviceI, float* weight,
      int width_pad, int starty, int startx, int patchSize) {

    int nBlocks = patchSize;
    int nThreadsPerBlock = patchSize;

    kernelInterpolatePatch<<<nBlocks, nThreadsPerBlock>>>(
        pDeviceRawDiff, pDeviceI, weight,
        width_pad, starty, startx, patchSize);

  }

  void normalizeMean(
      float* src, hipblasHandle_t handle, int patchSize) {

    int nBlocks = patchSize;
    int nThreadsPerBlock = patchSize;

    float mean;
    CUBLAS_CHECK (
        hipblasSasum(handle, patchSize * patchSize * 3, src, 1, &mean) );
    mean = mean / (patchSize * patchSize * 3);

    kernelNormalizeMean<<<nBlocks, nThreadsPerBlock>>>(
        src, mean, patchSize);

  }

}
